#include <stdio.h>
#include <hip/hip_runtime.h>

// tugas 1: alokasi memori dan transfer dari device ke host

__global__ void
kern(int *A)
{
    	int idx = blockDim.x * blockIdx.x + threadIdx.x;
 	A[idx] = idx;
}

/**
 * Host main routine
 */
int   main(void)
{
	// alokasikan memori, dan salin nilainya
	dim3 grid,block;
	block.x = 4;
	grid.x = 12/block.x;
	//kern<<grid,block>>(d_A);
	// copy result
	return 0;
}
